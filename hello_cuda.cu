/*
    Hello World - CUDA

    Compile by:
        nvcc hello_cuda.c -o hello_cuda
*/



#include <hip/hip_runtime.h>
#include <stdio.h>


// Global specifier indicates that the function runs on device (GPU).
// These functions can be called through the host code (e.g. main()) and also known as "kernels".
__global__ void cuda_hello(){
        printf("Hello World from GPU!\n");
}


int main(){
        // When kernel is called, it's execution configuration is provided through <<<...>>> syntax.
        // This is a "kernel launch".
        cuda_hello<<<1, 1>>>();
        return 0;
}
